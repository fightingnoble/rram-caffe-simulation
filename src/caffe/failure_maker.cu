#include "hip/hip_runtime.h"
#include "caffe/util/device_alternate.hpp"
#include "caffe/failure_maker.hpp"

namespace caffe {
  template <typename Dtype>
  __global__ void FailKernel(const int n, Dtype* iters, Dtype* data, const Dtype* diff) {//Dtype* values
    Dtype epsilon = 1e-20;
    CUDA_KERNEL_LOOP(index, n) {
      if (iters[index] <= 0) {
	// this cell is already broken
	//data[index] = values[index];
	data[index] = 0;//values[index];
      } else {
	// strategy1: not update when gradient is too small
	if (diff[index] < epsilon && diff[index] > -epsilon) {
	  continue;
	}
	iters[index] -= 100; // batch size. FIXME: how to make this exp more general
	if (iters[index] <= 0) {
	  //values[index] = 0;
	  data[index] = 0;
	}
      }
    }
  }

  template <typename Dtype>
  void GaussianFailureMaker<Dtype>::Fail_gpu(int iter) {
    for (int i = 0; i < fail_iterations_.size(); i++) {
      int count = fail_iterations_[i]->count();
      FailKernel<Dtype><<<CAFFE_GET_BLOCKS(count),
	CAFFE_CUDA_NUM_THREADS>>>(count, 
				  fail_iterations_[i]->mutable_gpu_data(),
				  //fail_iterations_[i]->mutable_gpu_diff();
				  this->net_->failure_learnable_params()[i]->mutable_gpu_data(),
				  this->net_->failure_learnable_params()[i]->gpu_diff());
    }
  }
  
  template void GaussianFailureMaker<double>::Fail_gpu(int);
  template void GaussianFailureMaker<float>::Fail_gpu(int);
}
