#include "hip/hip_runtime.h"
#include "caffe/util/device_alternate.hpp"
#include "caffe/failure_maker.hpp"

namespace caffe {
  template <typename Dtype>
  __global__ void FailureThresholdKernel(const int n, Dtype* values, Dtype split1, Dtype split2) {
    CUDA_KERNEL_LOOP(index, n) {
      if (values[index] < split1) {
	values[index] = -1;
      } else if (values[index] < split2) {
	values[index] = 0;
      } else {
	values[index] = 1;
      }
    }
  }

  template <typename Dtype>
  void failure_threshold(const int n, Dtype* values, Dtype split1, Dtype split2) {
    FailureThresholdKernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, values, split1, split2);
  }

  template <typename Dtype>
  __global__ void FailKernel(const int n, Dtype* iters, const Dtype* values, Dtype* data, const Dtype* diff) {
    Dtype epsilon = 1e-20;
    CUDA_KERNEL_LOOP(index, n) {
      if (iters[index] <= 0) {
	// this cell is already broken
	data[index] = values[index];
      } else {
	// strategy1: not update when gradient is too small
	if (diff[index] < epsilon && diff[index] > -epsilon) {
	  continue;
	}
	iters[index] -= 100; // batch size. FIXME: how to make this exp more general
	if (iters[index] <= 0) {
	  data[index] = values[index];
	}
      }
    }
  }

  template <typename Dtype>
  void GaussianFailureMaker<Dtype>::Fail_gpu(int iter) {
    for (int i = 0; i < fail_iterations_.size(); i++) {
      int count = fail_iterations_[i]->count();
      int N = CAFFE_GET_BLOCKS(count);
      // hiprandState_t* states;
      // hipMalloc((void**) &states, N * sizeof(hiprandState_t));
      // InitRandom<<<N, 1>>>(time(NULL), states);

      FailKernel<Dtype><<<N, CAFFE_CUDA_NUM_THREADS>>>(count, 
						       fail_iterations_[i]->mutable_gpu_data(),
						       fail_iterations_[i]->mutable_gpu_diff(),
						       this->net_->failure_learnable_params()[i]->mutable_gpu_data(),
						       this->net_->failure_learnable_params()[i]->gpu_diff());
	}
  }
  
  template void GaussianFailureMaker<double>::Fail_gpu(int);
  template void GaussianFailureMaker<float>::Fail_gpu(int);

  template void failure_threshold<float>(const int n, float* values, float split1, float split2);
  template void failure_threshold<double>(const int n, double* values, double split1, double split2);
}
